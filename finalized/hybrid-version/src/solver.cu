#include "hip/hip_runtime.h"
#include "../include/solver.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stddef.h>

/**
 * @brief CUDA kernel to advance the solution by one time step using the finite difference method.
 */
__global__ void heat_update(float *u, float *u_new, int nx, int ny, float alpha) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i > 0 && i < nx-1 && j > 0 && j < ny-1) {
        int idx = j*nx + i;
        u_new[idx] = u[idx] + alpha * (
            u[(j+1)*nx + i] + u[(j-1)*nx + i] +
            u[j*nx + (i+1)] + u[j*nx + (i-1)] - 4*u[idx]
        );
    }
}

/**
 * @brief Solve the heat equation using a hybrid CUDA+OpenMP approach.
 *
 * Advances the solution for a given number of timesteps on the device, using CUDA for computation.
 */
void solve_heat_equation(float *u, int nx, int ny, int timesteps, float alpha) {
    if (!u || nx < 2 || ny < 2 || timesteps < 1) return;
    float *d_u = NULL, *d_u_new = NULL;
    size_t size = nx * ny * sizeof(float);
    hipMalloc(&d_u, size);
    hipMalloc(&d_u_new, size);
    hipMemcpy(d_u, u, size, hipMemcpyHostToDevice);
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((nx+15)/16, (ny+15)/16);
    for (int t = 0; t < timesteps; ++t) {
        heat_update<<<numBlocks, threadsPerBlock>>>(d_u, d_u_new, nx, ny, alpha);
        hipDeviceSynchronize();
        float *tmp = d_u;
        d_u = d_u_new;
        d_u_new = tmp;
    }
    hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost);
    hipFree(d_u);
    hipFree(d_u_new);
} 